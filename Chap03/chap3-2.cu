#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

void checkDeviceMemory(void) {
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("Device memory (free/total) = %lld%lld bytes\n", free, total);
}

int main(void) {
    int* dDataPtr;
    hipError_t errCode;

    checkDeviceMemory();
    errCode = hipMalloc(&dDataPtr, sizeof(int) * 1024 * 1024);
    printf("hipMalloc - %s\n", hipGetErrorName(errCode));
    checkDeviceMemory();

    errCode = hipMemset(dDataPtr, 0, sizeof(int) * 1024 * 1024);
    printf("hipMemset - %s\n", hipGetErrorName(errCode);

    errCode = hipFree(dDataPtr);
    printf("hipFree - %s\n", hipGetErrorName(errCode));
    checkDeviceMemory();

    return 0;
}