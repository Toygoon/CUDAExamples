#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void printData(int*);
__global__ void setData(int*);

int main(void) {
    int data[10] = {0};
    for (int i = 0; i < 10; i++) {
        data[i] = 1;
    }

    int* dDataPtr;
    hipMalloc(&dDataPtr, sizeof(int) * 10);
    hipMemset(dDataPtr, 0, sizeof(int) * 10);

    printf("Data in device: ");
    printData<<<1, 10>>>(dDataPtr);

    hipMemcpy(dDataPtr, data, sizeof(int) * 10, hipMemcpyHostToDevice);
    printf("\nHost -> Device: ");
    printData<<<1, 10>>>(dDataPtr);

    hipMemcpy(data, dDataPtr, sizeof(int) * 10, hipMemcpyDeviceToHost);
    printf("\nDevice -> Host: ");

    for (int i = 0; i < 10; i++) {
        printf("%d", data[i]);
    }

    return 0;
}

__global__ void printData(int* _dDataPtr) {
    printf("%d", _dDataPtr[threadIdx.x]);
}

__global__ void setData(int* _dDataPtr) {
    _dDataPtr[threadIdx.x] = 2;
}