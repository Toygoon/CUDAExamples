#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"


// The size of the vector
#define NUM_DATA 33554432

__global__ void vecAdd(int*, int*, int*);
double getElapsedTime(clock_t);

int main(void) {
    // Vectors on the host : a, b, c, hc
    // Vectors on the device : da, db, dc
    int *a, *b, *c, *hc, *da, *db, *dc, memSize = sizeof(int) * NUM_DATA;
    clock_t start;
    printf("%d elements, memSize = %d bytes\n", NUM_DATA, memSize);

    // Memory allocation on the host-side
    a = new int[NUM_DATA];
    memset(a, 0, memSize);
    b = new int[NUM_DATA];
    memset(b, 0, memSize);
    c = new int[NUM_DATA];
    memset(c, 0, memSize);
    hc = new int[NUM_DATA];
    memset(hc, 0, memSize);

    // Data generation
    for (int i = 0; i < NUM_DATA; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    // Vector sum on host (for performance comparison)
    start = clock();
    for (int i = 0; i < NUM_DATA; i++) {
        hc[i] = a[i] + b[i];
    }
    printf("Vector sum on host : %.2f sec\n", getElapsedTime(start));

    // Memory allocation on the device-side
    hipMalloc(&da, memSize);
    hipMemset(da, 0, memSize);
    hipMalloc(&db, memSize);
    hipMemset(db, 0, memSize);
    hipMalloc(&dc, memSize);
    hipMemset(dc, 0, memSize);

    // Data copy : Host -> Device
    hipMemcpy(da, a, memSize, hipMemcpyHostToDevice);
    hipMemcpy(db, b, memSize, hipMemcpyHostToDevice);

    start = clock();
    // Kernel call
    vecAdd<<<1, NUM_DATA>>>(da, db, dc);
    printf("Vector sum on GPU : %.2f sec\n", getElapsedTime(start));

    // Copy results : Device -> Host
    hipMemcpy(c, dc, memSize, hipMemcpyDeviceToHost);

    // Release device memory
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    // Release host memory
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}

// Simple vector sum kernel
__global__ void vecAdd(int* _a, int* _b, int* _c) {
    int id = threadIdx.x;
    _c[id] = _a[id] + _b[id];
}

// Get elapsed time to compare the calculating duration
double getElapsedTime(clock_t start) {
    return ((double)(clock() - start)) / CLOCKS_PER_SEC;
}