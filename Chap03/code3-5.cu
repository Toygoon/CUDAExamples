#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "hip/hip_runtime.h"


// The size of the vector
#define NUM_DATA 1024

__global__ void vecAdd(int*, int*, int*);

int main(void) {
    // Vectors on the host : a, b, c, hc
    // Vectors on the device : da, db, dc
    int *a, *b, *c, *hc, *da, *db, *dc, memSize = sizeof(int) * NUM_DATA;
    printf("%d elements, memSize = %d bytes\n", NUM_DATA, memSize);

    // Memory allocation on the host-side
    a = new int[NUM_DATA];
    memset(a, 0, memSize);
    b = new int[NUM_DATA];
    memset(b, 0, memSize);
    c = new int[NUM_DATA];
    memset(c, 0, memSize);
    hc = new int[NUM_DATA];
    memset(hc, 0, memSize);

    // Data generation
    for (int i = 0; i < NUM_DATA; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    // Vector sum on host (for performance comparison)
    for (int i = 0; i < NUM_DATA; i++) {
        hc[i] = a[i] + b[i];
    }

    // Memory allocation on the device-side
    hipMalloc(&da, memSize);
    hipMemset(da, 0, memSize);
    hipMalloc(&db, memSize);
    hipMemset(db, 0, memSize);
    hipMalloc(&dc, memSize);
    hipMemset(dc, 0, memSize);

    // Data copy : Host -> Device
    hipMemcpy(da, a, memSize, hipMemcpyHostToDevice);
    hipMemcpy(db, b, memSize, hipMemcpyHostToDevice);

    // Kernel call
    vecAdd<<<1, NUM_DATA>>>(da, db, dc);

    // Copy results : Device -> Host
    hipMemcpy(c, dc, memSize, hipMemcpyDeviceToHost);

    // Release device memory
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    // Check results
    bool result = true;
    for (int i = 0; i < NUM_DATA; i++) {
        if (hc[i] != c[i]) {
            printf("[%d] The result is not matched! (%d, %d)\n", i, hc[i], c[i]);

            result = false;
        }
    }

    if (result)
        printf("GPU works well!\n");

    // Release host memory
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}

// Simple vector sum kernel
__global__ void vecAdd(int* _a, int* _b, int* _c) {
    int id = threadIdx.x;
    _c[id] = _a[id] + _b[id];
}