#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void helloCuda(void) {
    printf("Hello CUDA from GPU.\n");
}

int main(void) {
    printf("Hello GPU from CPU.\n");
    helloCuda<<<1, 10>>>();

    return 0;
}